#include "hip/hip_runtime.h"
#include <algorithm>
#include <bitset>
#include <bit>
#include <cstdlib>
#include <cstdint>
#include <iostream>
#include <pthread.h>
#include <stdio.h>
#include <thread>
#include "data_generator.cuh"
#include <vector>

// #define DISABLE_CUDA_TIME
#include "cuda_time.cuh"
#include "cuda_try.cuh"

#include "csv_loader.hpp"
#include "utils.cuh"
#include "data_generator.cuh"
#include "benchmarks.cuh"
#include "kernels/data_generator.cuh"
#include <unistd.h>

typedef float input_data_type;
static float threshold = 200;
bool predicate_function(input_data_type f)
{
    return f > threshold;
}
int main(int argc, char** argv)
{
    int lines = 0;
    const char* csv_path = "../res/Arade_1.csv";
    int iterations = 100;
    bool report_failures = false;

    int grid_size_max = 2048;
    int grid_size_min = 32;
    bool use_csv = false;
    bool use_pattern_mask = true;
    int pattern_length = 8;
    uint32_t pattern;
    float selectivity = 0.5;
    bool use_uniform = false;
    bool use_zipf = false;
    int option;
    while ((option = getopt(argc, argv, ":zurd:l:i:f:p:s:t:g:m:")) != -1) {
        switch (option) {
            case 'g': {
                int grid_size_max = atoi(optarg);
                fprintf(stderr, "using max grid size %i\n", grid_size_max);
            } break;
            case 'm': {
                int grid_size_min = atoi(optarg);
                fprintf(stderr, "using min grid size %i\n", grid_size_min);
                if (std::popcount((uint32_t)grid_size_min) != 1) {
                    error("min grid size has to be a power of two\n");
                }
            } break;
            case 'd': {
                int device = atoi(optarg);
                fprintf(stderr, "setting device numer to %i\n", device);
                CUDA_TRY(hipSetDevice(device));
            } break;
            case 'l': {
                lines = atoi(optarg);
                fprintf(stderr, "setting line count to %i\n", lines);
            } break;
            case 'i': {
                iterations = atoi(optarg);
                fprintf(stderr, "setting iteration count to %i\n", iterations);
            } break;
            case 'r': {
                fprintf(stderr, "will report failures\n");
                report_failures = true;
            } break;
            case 'f': {
                csv_path = optarg;
                lines = 0;
            } break;
            case 'p': {
                pattern_length = atoi(optarg);
                if (pattern_length > 32 || pattern_length < 1) pattern_length = 8;
                fprintf(stderr, "setting pattern length to %i\n", pattern_length);
            } break;
            case 's': {
                selectivity = atof(optarg);
                fprintf(stderr, "setting selectivity to %f\n", selectivity);
            } break;
            case 't': {
                threshold = atof(optarg);
                fprintf(stderr, "setting value threshold to%f\n", threshold);
                use_csv = true;
            } break;
            case 'z': {
                fprintf(stderr, "using zipf mask\n");
                use_zipf = true;
            } break;
            case 'u': {
                fprintf(stderr, "using uniform mask\n");
                use_uniform = true;
            } break;
            case ':': {
                fprintf(stderr, "-%c needs a value\n", optopt);
                exit(-1);
            } break;
            case '?': { // used for some unknown options
                fprintf(stderr, "unknown option: %c\n", optopt);
                exit(-1);
            } break;
        }
    }
    if (use_zipf || use_uniform || use_csv) {
        use_pattern_mask = false;
    }
    if (use_zipf + use_uniform + use_csv + use_pattern_mask != 1) {
        error("can only use one mask type\n");
    }
    if (use_pattern_mask) {
        int pattern_one_count = pattern_length * selectivity;
        fast_prng rng(42);
        pattern = 0;
        while (pattern_one_count > 0) {
            int i = rng.rand() % pattern_length;
            if (((pattern >> (31 - i)) & 0x1) == 0) {
                pattern_one_count--;
                pattern |= 1 << (31 - i);
            }
        }
        // generate_mask_uniform((uint8_t*)&pattern, 0, 4, selectivity);
        // pattern = pattern << (32 - pattern_length);
        std::bitset<32> pattern_bitset(pattern); // load data
        std::stringstream ss;
        ss << pattern_bitset;
        std::cout << "pattern: " << ss.str().substr(0, pattern_length) << "\n";
    }
    std::vector<input_data_type> col;
    if (!use_csv) {
        fprintf(stderr, "generating %i lines of input\n", lines);
        col.resize(lines);
        generate_mask_uniform((uint8_t*)&col[0], 0, lines * 4, 0.5);
    }
    else {
        fprintf(stderr, "parsing %s\n", csv_path);
        load_csv(csv_path, {3}, col);
        if (lines > 0) {
            col.resize(std::min(col.size(), static_cast<size_t>(lines)));
        }
    }
    input_data_type* d_input = vector_to_gpu(col);
    input_data_type* d_output = alloc_gpu<input_data_type>(col.size() + 1);

    // gen predicate mask
    size_t one_count = 0;
    std::vector<uint8_t> pred;
    if (use_csv) {
        pred = gen_predicate(col, predicate_function, &one_count);
    }
    if (use_uniform) {
        pred.resize(ceildiv(col.size(), 8));
        generate_mask_uniform(&pred[0], 0, pred.size(), selectivity, &one_count);
    }
    if (use_zipf) {
        pred.resize(ceildiv(col.size(), 8));
        generate_mask_zipf(&pred[0], pred.size(), 0, pred.size(), &one_count);
    }
    if (use_pattern_mask) {
        pred.resize(ceildiv(col.size(), 8));
        // mask from pattern instead
        generate_mask_pattern(&pred[0], 0, pred.size(), pattern, pattern_length, &one_count);
    }
    // make sure unused bits in bitmask are 0
    int unused_bits = overlap(col.size(), 8);
    if (unused_bits) {
        pred.back() >>= unused_bits;
        pred.back() <<= unused_bits;
    }

    // put predicate mask on gpu
    uint8_t* d_mask = vector_to_gpu(pred);

    fprintf(stderr, "line count: %zu, one count: %zu, percentage: %f\n", col.size(), one_count, (double)one_count / col.size());

    // gen cpu side validation
    std::vector<input_data_type> validation;
    validation.resize(col.size());
    size_t out_length = generate_validation(&col[0], &pred[0], &validation[0], col.size());
    input_data_type* d_validation = vector_to_gpu(validation);

    fprintf(stderr, "starting benchmark\n");

    // prepare candidates for benchmark
    intermediate_data id{col.size(), 1024, 8}; // setup shared intermediate data

    std::vector<std::pair<std::string, std::function<timings(int, int)>>> benchs;

    benchs.emplace_back(
        "bench1_base_variant", [&](int bs, int gs) { return bench1_base_variant(&id, d_input, d_mask, d_output, col.size(), 1024, bs, gs); });
    benchs.emplace_back("bench2_base_variant_skipping", [&](int bs, int gs) {
        return bench2_base_variant_skipping(&id, d_input, d_mask, d_output, col.size(), 1024, bs, gs);
    });
    // benchs.emplace_back(
    //     "bench3_3pass_streaming", [&](int bs, int gs) { return bench3_3pass_streaming(&id, d_input, d_mask, d_output, col.size(), 1024, bs, gs);
    //     });
    benchs.emplace_back("bench4_3pass_optimized_read_non_skipping_cub_pss", [&](int bs, int gs) {
        return bench4_3pass_optimized_read_non_skipping_cub_pss(&id, d_input, d_mask, d_output, col.size(), 1024, bs, gs);
    });
    benchs.emplace_back("bench5_3pass_optimized_read_skipping_partial_pss", [&](int bs, int gs) {
        return bench5_3pass_optimized_read_skipping_partial_pss(&id, d_input, d_mask, d_output, col.size(), 1024, bs, gs);
    });
    benchs.emplace_back("bench6_3pass_optimized_read_skipping_two_phase_pss", [&](int bs, int gs) {
        return bench6_3pass_optimized_read_skipping_two_phase_pss(&id, d_input, d_mask, d_output, col.size(), 1024, bs, gs);
    });
    benchs.emplace_back("bench7_3pass_optimized_read_skipping_cub_pss", [&](int bs, int gs) {
        return bench7_3pass_optimized_read_skipping_cub_pss(&id, d_input, d_mask, d_output, col.size(), 1024, bs, gs);
    });
    benchs.emplace_back("bench8_cub_flagged", [&](int bs, int gs) { return bench8_cub_flagged(&id, d_input, d_mask, d_output, col.size()); });

    if (use_pattern_mask) {
        benchs.emplace_back("bench9_pattern", [&](int bs, int gs) {
            return bench9_pattern(&id, d_input, pattern, pattern_length, d_output, col.size(), 1024, bs, gs);
        });
    }

    std::cout << "benchmark;block_size;grid_size;time_popc;time_pss1;time_pss2;time_proc;time_total" << std::endl;
    // run benchmark
    for (int grid_size = grid_size_min; grid_size <= grid_size_max; grid_size *= 2) {
        for (int block_size = 32; block_size <= 1024; block_size *= 2) {
            std::vector<timings> timings(benchs.size());
            for (int it = 0; it < iterations; it++) {
                for (size_t i = 0; i < benchs.size(); i++) {
                    timings[i] += benchs[i].second(block_size, grid_size);
                    size_t failure_count;
                    if (!validate(&id, d_validation, d_output, out_length, report_failures, &failure_count)) {
                        fprintf(
                            stderr, "validation failure in bench %s (%d, %d), run %i: %zu failures\n", benchs[i].first.c_str(), block_size, grid_size,
                            it, failure_count);
                        // exit(EXIT_FAILURE);
                    }
                }
            }
            for (int i = 0; i < benchs.size(); i++) {
                std::cout << benchs[i].first << ";" << block_size << ";" << grid_size << ";" << timings[i] / static_cast<float>(iterations)
                          << std::endl;
            }
        }
    }
    return 0;
}
