#include <bitset>
#include <cstdlib>
#include <hipcub/hipcub.hpp>
#include <cstdint>
#include <iostream>
#include <stdio.h>
#include <thread>
#include <vector>

#include "cuda_time.cuh"
#include "cuda_try.cuh"
#include "data_generator.cuh"

#include "fast_prng.cuh"
#include "kernels/data_generator.cuh"

#include "csv_loader.hpp"
#include "utils.hpp"

#include <src/cub_wraps.cuh>



int gen_dummy_data(){
    int cuda_dev_id = 0;
    CUDA_TRY(hipSetDevice(cuda_dev_id));

    uint64_t cnt_b = 1<<30; // mask size in bytes
    uint64_t cnt_e = cnt_b*8; // element count

    uint8_t* d_mask;
    CUDA_TRY(hipMalloc(&d_mask, sizeof(uint8_t)*cnt_b));
    printf("GPU generation...");
    kernel_generate_mask_uniform<<<64, 32>>>(d_mask, cnt_b, 0.5);
    CUDA_TRY(hipDeviceSynchronize());
    printf("done\n");

    uint8_t* d_mask2;
    CUDA_TRY(hipMalloc(&d_mask2, sizeof(uint8_t)*cnt_b));
    CUDA_TRY(hipMemcpy(d_mask2, d_mask, sizeof(uint8_t)*cnt_b, hipMemcpyDeviceToDevice));

    uint64_t* d_failure_count;
    CUDA_TRY(hipMalloc(&d_failure_count, sizeof(uint64_t)));
    CUDA_TRY(hipMemset(d_failure_count, 0x00, sizeof(uint64_t)));

    // introduce some artificial errors
    fast_prng rng(17);
    for (int i = 0; i < 20; i++) {
        uint64_t rand_idx = rng.rand() % cnt_b;
        CUDA_TRY(hipMemset(d_mask2+rand_idx, 0x00, sizeof(uint8_t)));
    }

    printf("GPU check validation...");
    kernel_check_validation<<<64, 32>>>(d_mask, d_mask2, cnt_b, d_failure_count);
    CUDA_TRY(hipDeviceSynchronize());
    printf("done\n");

    uint64_t h_failure_count = 0;
    CUDA_TRY(hipMemcpy(&h_failure_count, d_failure_count, sizeof(uint64_t), hipMemcpyDeviceToHost));
    printf("validation failures = %lu\n", h_failure_count);

    CUDA_TRY(hipFree(d_failure_count));
    CUDA_TRY(hipFree(d_mask2));
    CUDA_TRY(hipFree(d_mask));

    printf("done\n");
    return 0;
}

int main(int argc, char** argv)
{
    //load data
    std::vector<float> col;
    load_csv("../res/Arade_1.csv", {3}, col);
    float* d_input = vector_to_gpu(col);
    float* d_output = alloc_gpu<float>(col.size());
    // gen predicate mask
    auto pred = gen_predicate(col, +[](float f){return f > 200;});
    uint8_t* d_mask = vector_to_gpu(pred);
    uint32_t* d_selected_out = alloc_gpu<uint32_t>(1);
    uint64_t* d_failure_count = alloc_gpu<uint64_t>(1);
    // run cub
    hipEvent_t start, end;
    CUDA_TRY(hipEventCreate(&start));
    CUDA_TRY(hipEventCreate(&end));
    launch_cub_flagged_biterator(start, end, d_input, d_output, d_mask, d_selected_out, col.size()); 
    //gen cpu side validation
    std::vector<float> validation;
    validation.resize(col.size());
    size_t out_length = generate_validation(&col[0], &pred[0], &validation[0], col.size());
    float* d_validation = vector_to_gpu(validation);
    
    // cross check validation
    kernel_check_validation<<<64, 32>>>(d_validation, d_output, out_length, d_failure_count);
    auto vec = gpu_to_vector(d_failure_count, 1);
    std::cout << vec[0] << std::endl;
    return 0;
   
}
