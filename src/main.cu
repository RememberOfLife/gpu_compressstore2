#include <bitset>
#include <cstdlib>
#include <hipcub/hipcub.hpp>
#include <cstdint>
#include <iostream>
#include <stdio.h>
#include <thread>
#include <vector>

#include "cuda_time.cuh"
#include "cuda_try.cuh"
#include "data_generator.cuh"

#include "fast_prng.cuh"
#include "kernels/data_generator.cuh"

template <typename T>
void cpu_buffer_print(T* h_buffer, uint32_t offset, uint32_t length)
{
    for (int i = offset; i < offset+length; i++) {
        std::bitset<sizeof(T)*8> bits(h_buffer[i]);
        std::cout << bits << " - " << unsigned(h_buffer[i]) << "\n";
    }
}

template <typename T>
void gpu_buffer_print(T* d_buffer, uint32_t offset, uint32_t length)
{
    T* h_buffer = static_cast<T*>(malloc(length*sizeof(T)));
    CUDA_TRY(hipMemcpy(h_buffer, d_buffer+offset, length*sizeof(T), hipMemcpyDeviceToHost));
    for (int i = 0; i < length; i++) {
        std::bitset<sizeof(T)*8> bits(h_buffer[i]);
        std::cout << bits << " - " << unsigned(h_buffer[i]) << "\n";
    }
    free(h_buffer);
}

int main()
{
    int cuda_dev_id = 0;
    CUDA_TRY(hipSetDevice(cuda_dev_id));

    uint64_t cnt_b = 1<<30; // mask size in bytes
    uint64_t cnt_e = cnt_b*8; // element count

    uint8_t* d_mask;
    CUDA_TRY(hipMalloc(&d_mask, sizeof(uint8_t)*cnt_b));
    printf("GPU generation...");
    kernel_generate_mask_uniform<<<64, 32>>>(d_mask, cnt_b, 0.5);
    CUDA_TRY(hipDeviceSynchronize());
    printf("done\n");

    uint8_t* d_mask2;
    CUDA_TRY(hipMalloc(&d_mask2, sizeof(uint8_t)*cnt_b));
    CUDA_TRY(hipMemcpy(d_mask2, d_mask, sizeof(uint8_t)*cnt_b, hipMemcpyDeviceToDevice));

    uint64_t* d_failure_count;
    CUDA_TRY(hipMalloc(&d_failure_count, sizeof(uint64_t)));
    CUDA_TRY(hipMemset(d_failure_count, 0x00, sizeof(uint64_t)));

    // introduce some artificial errors
    fast_prng rng(17);
    for (int i = 0; i < 20; i++) {
        uint64_t rand_idx = rng.rand() % cnt_b;
        CUDA_TRY(hipMemset(d_mask2+rand_idx, 0x00, sizeof(uint8_t)));
    }

    printf("GPU check validation...");
    kernel_check_validation<<<64, 32>>>(d_mask, d_mask2, cnt_b, d_failure_count);
    CUDA_TRY(hipDeviceSynchronize());
    printf("done\n");

    uint64_t h_failure_count = 0;
    CUDA_TRY(hipMemcpy(&h_failure_count, d_failure_count, sizeof(uint64_t), hipMemcpyDeviceToHost));
    printf("validation failures = %lu\n", h_failure_count);

    CUDA_TRY(hipFree(d_failure_count));
    CUDA_TRY(hipFree(d_mask2));
    CUDA_TRY(hipFree(d_mask));

    printf("done\n");
    return 0;
}
